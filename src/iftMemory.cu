#include "hip/hip_runtime.h"
#include "iftMemory.cuh"

#include "ift/core/tools/Dialog.h"
// these includes CANNOT be inside an extern "C" {
#include <hip/hip_runtime.h>
#include <hipblas.h>


/* device=0, 1, etc */

void iftStartGPU(int device) {
    int nDevices;
    hipError_t err = hipGetDeviceCount(&nDevices);
    if (err != hipSuccess) printf("%s\n", hipGetErrorString(err));
    if ((device > nDevices-1) || (device < 0))
        iftError("Invalid device number","iftStartGPU");	

    err = hipSetDevice(device);
    if (err != hipSuccess) iftError("Check Cuda documentation for error: %s.\n", "iftStartGPU", err);
    
    cublasInit();
    struct hipDeviceProp_t props;
    err = hipGetDeviceProperties(&props, device);
    if (err != hipSuccess) iftError("Check Cuda documentation for error: %s.\n", "iftStartGPU", err);

    printf("GPU-%d: %s @ %.0fMHz. %.0fGB/%.0fKB Global/Shared memory.\n\n", device, props.name, (float)props.clockRate/1000.0, (float)props.totalGlobalMem/pow(1024.0,3), props.sharedMemPerBlock/1024.0);

    size_t free, total;

    err = hipMemGetInfo(&free, &total);
    if (err != hipSuccess) {
        printf("Error hipMemGetInfo\n");
    } else {
        printf("Free=%lu, total=%lu\n", free, total);
    }
}

void iftStopGPU() {
    hipDeviceReset();
}

void iftCopyToGPU(void* dst, void* src, size_t size) {
    hipError_t status = hipMemcpy(dst, src, size, hipMemcpyHostToDevice);
    if(status != hipSuccess) {
        iftError("Check Cuda documentation for error %d.\n", "iftCopyToGPU", status);
    }
}

void iftCopyFromGPU(void* dst, void* src, size_t size) {
    hipError_t status = hipMemcpy(dst, src, size, hipMemcpyDeviceToHost);
    if(status != hipSuccess) {
        iftError("Check Cuda documentation for error %d.\n", "iftCopyFromGPU", status);
    }
}

void iftCopyIntArrayFromGPU(int* dst, int* src, int n) {
    iftCopyFromGPU((void*)dst, (void*)src, n*sizeof(int));
}

void iftCopyIntArrayToGPU(int* dst, int* src, int n) {
    iftCopyToGPU((void*)dst, (void*)src, n*sizeof(int));
}

int* iftAllocIntArrayGPU(int n) {
    return (int*) iftAllocGPU(n, sizeof(int));
}

void iftCopyFloatArrayFromGPU(float* dst, float* src, int n) {
    iftCopyFromGPU((void*)dst, (void*)src, n*sizeof(float));
}

void iftCopyFloatArrayToGPU(float* dst, float* src, int n) {
    iftCopyToGPU((void*)dst, (void*)src, n*sizeof(float));
}

float* iftAllocFloatArrayGPU(int n) {
    return (float*) iftAllocGPU(n, sizeof(float));
}

void *iftAllocGPU(int n, size_t size) {
    void *mem = NULL;
    
    hipError_t status = hipMalloc(&mem, n * size);
    if(status==hipErrorOutOfMemory)//remove if slow down the function
    {
        iftError("Could not allocate enough memory.", "iftAllocGPU");
    }
    else if(status!=hipSuccess) {
        iftError("Check Cuda documentation for error %d.\n", "iftAllocGPU", status);
    }

    return mem;
}

void iftFreeGPU(void *mem) {
    hipFree(mem);
}

float iftGetFreeMemoryGPU(int device)
{
    size_t freeMem, totalMem;
    hipError_t err = hipMemGetInfo(&freeMem, &totalMem);
    if (err != hipSuccess)
        printf("%s\n", hipGetErrorString(err));

    return freeMem;
}


