#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include "hip/hip_runtime.h"


/* ----------------------------------------------------------------------------------------------- */

void get_device_memory(double* free_db, double* used_db, double* total_db) {

	// gets memory usage in byte
	size_t free_byte;
	size_t total_byte;
	hipError_t cuda_status = hipMemGetInfo(&free_byte, &total_byte);
	if (hipSuccess != cuda_status){
		printf("Error: hipMemGetInfo fails, %s \n", hipGetErrorString(cuda_status));
		exit(EXIT_FAILURE);
	}

	*free_db = (double)free_byte;
	*total_db = (double)total_byte;
	*used_db = *total_db - *free_db;

	return;
}


void print_device_memory(){
	double free_db, used_db, total_db;
	get_device_memory(&free_db, &used_db, &total_db);

	printf("GPU memory usage: used = %f MB, free = %f MB, total = %f MB\n",
		used_db / 1024.0 / 1024.0, free_db / 1024.0 / 1024.0, total_db / 1024.0 / 1024.0);
}
/* ----------------------------------------------------------------------------------------------- */

void exit_on_error(char* info) {
	printf("\nERROR: %s\n", info);
	fflush(stdout);

	//free(info);
	exit(EXIT_FAILURE);
	return;
}

/* ----------------------------------------------------------------------------------------------- */

void exit_on_cuda_error(char* kernel_name) {
	// sync and check to catch errors from previous async operations
	hipDeviceSynchronize();
	hipError_t err = hipGetLastError();
	if (err != hipSuccess){
		printf("Error after %s: %s\n", kernel_name, hipGetErrorString(err));

		// releases previous contexts
#if CUDA_VERSION < 4000
		hipDeviceReset();
#else
		hipDeviceReset();
#endif

		// stops program
		//free(kernel_name);
		exit(EXIT_FAILURE);
	}
}



/* ----------------------------------------------------------------------------------------------- */

// GPU initialization

/* ----------------------------------------------------------------------------------------------- */


void initialize_cuda_device(int* ncuda_devices, int verbose) {

	int device;
	int device_count;
	int myrank = 0;

	/*
	// cuda initialization (needs -lcuda library)
	// note:   hipInit initializes the driver API.
	//             it is needed for any following CUDA driver API function call (format cuFUNCTION(..) )
	//             however, for the CUDA runtime API functions (format cudaFUNCTION(..) )
	//             the initialization is implicit, thus hipInit() here would not be needed...
	hipError_t status = hipInit(0);
	if ( hipSuccess != status ) exit_on_error("CUDA driver API device initialization failed\n");

	// returns a handle to the first cuda compute device
	hipDevice_t dev;
	status = hipDeviceGet(&dev, 0);
	if ( hipSuccess != status ) exit_on_error("CUDA device not found\n");

	// gets device properties
	int major,minor;
	status = hipDeviceComputeCapability(&major,&minor,dev);
	if ( hipSuccess != status ) exit_on_error("CUDA device information not found\n");

	// make sure that the device has compute capability >= 1.3
	if (major < 1){
	fprintf(stderr,"Compute capability major number should be at least 1, got: %d \nexiting...\n",major);
	exit_on_error("CUDA Compute capability major number should be at least 1\n");
	}
	if (major == 1 && minor < 3){
	fprintf(stderr,"Compute capability should be at least 1.3, got: %d.%d \nexiting...\n",major,minor);
	exit_on_error("CUDA Compute capability major number should be at least 1.3\n");
	}
	*/

	// note: from here on we use the runtime API  ...

	// Gets number of GPU devices
	device_count = 0;
	hipGetDeviceCount(&device_count);
	char bufferError1[200] = "CUDA runtime error: hipGetDeviceCount failed\ncheck if driver and runtime libraries work together\nexiting...\n";
	exit_on_cuda_error(bufferError1);

	char bufferError2[200] = "CUDA runtime error: there is no device supporting CUDA\n";
	// returns device count to fortran
	if (device_count == 0) exit_on_error(bufferError2);
	*ncuda_devices = device_count;

	// Sets the active device
	if (device_count >= 1) {
		// generalized for more GPUs per node
		// note: without previous context release, hipSetDevice will complain with the cuda error
		//         "setting the device when a process is active is not allowed"

		// releases previous contexts
		#if CUDA_VERSION < 4000
				hipDeviceReset();
		#else
				hipDeviceReset();
		#endif

		// sets active device
		device = myrank % device_count;
		hipSetDevice(device);
		char bufferError3[200] = "hipSetDevice has invalid device";
		exit_on_cuda_error(bufferError3);

		// double check that device was  properly selected
		hipGetDevice(&device);
		if (device != (myrank % device_count)){
			printf("error rank: %d devices: %d \n", myrank, device_count);
			printf("  hipSetDevice()=%d\n  hipGetDevice()=%d\n", myrank%device_count, device);
			char bufferError4[200] = "CUDA set/get device error: device id conflict \n";
			exit_on_error(bufferError4);
		}
	}

	// returns a handle to the active device
	hipGetDevice(&device);

	// get device properties
	struct hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, device);

	// exit if the machine has no CUDA-enabled device
	if (deviceProp.major == 9999 && deviceProp.minor == 9999){
		printf("No CUDA-enabled device found, exiting...\n\n");
		char bufferError5[200] = "CUDA runtime error: there is no CUDA-enabled device found\n";
		exit_on_error(bufferError5);
	}

	// outputs device infos to file
	if (verbose>1){
		//printf("GPU device for rank: %d\n\n", myrank);
		printf("\n");
		// display device properties
		printf("Device Name = %s\n", deviceProp.name);
		printf("multiProcessorCount: %d\n", deviceProp.multiProcessorCount);
		printf("totalGlobalMem (in MB): %f\n", (unsigned long)deviceProp.totalGlobalMem / (1024.f * 1024.f));
		printf("totalGlobalMem (in GB): %f\n", (unsigned long)deviceProp.totalGlobalMem / (1024.f * 1024.f * 1024.f));
		printf("sharedMemPerBlock (in bytes): %lu\n", (unsigned long)deviceProp.sharedMemPerBlock);
		printf("Maximum number of threads per block: %d\n", deviceProp.maxThreadsPerBlock);
		printf("Maximum size of each dimension of a block: %d x %d x %d\n",
			deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]);
		printf("Maximum sizes of each dimension of a grid: %d x %d x %d\n",
			deviceProp.maxGridSize[0], deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]);
		printf("Compute capability of the device = %d.%d\n", deviceProp.major, deviceProp.minor);
		if (deviceProp.canMapHostMemory){
			printf("canMapHostMemory: TRUE\n");
		}
		else{
			printf("canMapHostMemory: FALSE\n");
		}
		if (deviceProp.deviceOverlap){
			printf("deviceOverlap: TRUE\n");
		}
		else{
			printf("deviceOverlap: FALSE\n");
		}

		// outputs initial memory infos via hipMemGetInfo()
		print_device_memory();
		printf("\n");
	}

	// make sure that the device has compute capability >= 1.3
	if (deviceProp.major < 1){
		printf("Compute capability major number should be at least 1, exiting...\n\n");
		char bufferError6[200] = "CUDA Compute capability major number should be at least 1\n";
		exit_on_error(bufferError6);
	}
	if (deviceProp.major == 1 && deviceProp.minor < 3){
		printf("Compute capability should be at least 1.3, exiting...\n");
		char bufferError7[200] = "CUDA Compute capability major number should be at least 1.3\n";
		exit_on_error(bufferError7);
	}
	// we use pinned memory for asynchronous copy
	if (!deviceProp.canMapHostMemory){
		printf("Device capability should allow to map host memory, exiting...\n");
		char bufferError8[200] = "Device capability should allow to map host memory, exiting...\n";
		exit_on_error(bufferError8);
	}
}